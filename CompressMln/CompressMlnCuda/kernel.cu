#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include "cudacaller.h"

__global__ void runKernel(const unsigned char* bits, int* bytes, const int sizeBits)
{
    int tid = threadIdx.x;
    int gid = blockDim.x * blockIdx.x;
    int i = gid + tid;

    if (i < sizeBits)
        atomicOr(&bytes[i / 8], ((bits[i] > 0) << (i % 8)));
}

bool cudaCaller::canCuda()
{
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        printf("No CUDA support device found");
        return false;
    }
    return true;
}

bool cudaCaller::doCuda(std::vector<unsigned char>& hostBits, std::vector<unsigned char>& hostBytes, int sizeBits)
{
    thrust::device_vector<unsigned char> deviceBits(hostBits.begin(), hostBits.end());
    thrust::device_vector<int> deviceBytes(hostBytes.size(), 0); // int!

    unsigned char* deviceBitsPtr = thrust::raw_pointer_cast(deviceBits.data());
    int* deviceBytesPtr = thrust::raw_pointer_cast(deviceBytes.data());

    int devNo = 0;
    hipDeviceProp_t iProp;
    hipGetDeviceProperties(&iProp, devNo);

    dim3 blockDim = iProp.maxThreadsPerBlock;
    dim3 gridDim = (sizeBits + blockDim.x - 1) / blockDim.x;

    hipError_t cudaStatus;
    runKernel << <gridDim, blockDim >> > (deviceBitsPtr, deviceBytesPtr, sizeBits);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching runKernel!\n", cudaStatus);
        return false;
    }

    thrust::copy(deviceBytes.begin(), deviceBytes.end(), hostBytes.begin());

    return true;
}

